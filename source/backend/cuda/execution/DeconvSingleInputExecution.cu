#include "hip/hip_runtime.h"
//
//  DeconvSingleInputExecution.cpp
//  MNN
//
//  Created by MNN on 2020/08/22.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "DeconvSingleInputExecution.hpp"

namespace MNN {
namespace CUDA {

template <typename T>
__global__ void cutPad(const size_t size, const T* input, const int old_height,
                    const int old_width, const int height, const int width, const int pad_top,
                    const int pad_left, T* output) {
    for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
        int block_num = pos / (width*height);
        int left = pos % (width*height);
        const int out_w = left % width;
        const int out_h = left / width % height;

        output[pos] = input[(block_num * old_height + out_h + pad_top) * old_width + out_w + pad_left];
    }
    return;
}

DeconvSingleInputExecution::DeconvSingleInputExecution(Backend* backend, const MNN::Op* op) : Execution(backend), mOp(op) {
    //MNN_PRINT("cuda DeconvSingleInput onInit in\n");
    auto conv       = op->main_as_Convolution2D();
    auto common     = conv->common();

    mKernelInfo.groups         = common->group();
    mKernelInfo.kernelX        = common->kernelX();
    mKernelInfo.kernelY        = common->kernelY();
    mKernelInfo.padMode        = common->padMode();
    mKernelInfo.padX           = common->padX();
    mKernelInfo.padY           = common->padY();

    if (nullptr != common->pads()) {
        mKernelInfo.padX = common->pads()->data()[1];
        mKernelInfo.padY = common->pads()->data()[0];
    }
    pad_left_  = mKernelInfo.padX;
    pad_right_ = mKernelInfo.padX;
    pad_top_ = mKernelInfo.padY;
    pad_bottom_ = mKernelInfo.padY;

    mKernelInfo.strideX        = common->strideX();
    mKernelInfo.strideY        = common->strideY();
    mKernelInfo.dilateX        = common->dilateX();
    mKernelInfo.dilateY        = common->dilateY();
    mKernelInfo.activationType = common->relu() ? 1 : (common->relu6() ? 2 : 0);

    use_relu_ = (mKernelInfo.activationType == 1);
    use_relu6_ = (mKernelInfo.activationType == 2);

    cudnn_handle_ = nullptr;
    input_desc_ = nullptr;
    output_desc_ = nullptr;
    filter_desc_ = nullptr;
    conv_desc_ = nullptr;
    padded_desc_ = nullptr;
    cudnn_data_type_ = HIPDNN_DATA_FLOAT;
    cudnn_data_type_len_ = 0;

    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    cudnn_handle_ = runtime->cudnn_handle();
    cudnn_check(hipdnnCreateTensorDescriptor(&input_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&output_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&padded_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&bias_desc_));
    cudnn_check(hipdnnCreateFilterDescriptor(&filter_desc_));
    cudnn_check(hipdnnCreateConvolutionDescriptor(&conv_desc_));
    cudnn_check(hipdnnCreateActivationDescriptor(&act_desc_));


    //weight host->device
    const float* filterDataPtr = nullptr;
    int weightSize = 0;
    std::shared_ptr<ConvolutionCommon::Int8Common> quanCommon;
    ConvolutionCommon::getConvParameters(&quanCommon, conv, &filterDataPtr, &weightSize);
    weightTensor.reset(Tensor::createDevice<float>({weightSize}));
    backend->onAcquireBuffer(weightTensor.get(), Backend::STATIC);
    mFilter = (void *)weightTensor.get()->buffer().device;
    cuda_check(hipMemcpy(mFilter, filterDataPtr, weightSize*sizeof(float), hipMemcpyHostToDevice));


    if(conv->bias()->size() != 0) {
        int biasSize = conv->bias()->size();
        biasTensor.reset(Tensor::createDevice<float>({biasSize}));
        backend->onAcquireBuffer(biasTensor.get(), Backend::STATIC);
        mBias = (void *)biasTensor.get()->buffer().device;

        cuda_check(hipMemcpy(mBias, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));
        
        int bias_size = conv->bias()->size();
        int dim_bias[] = {1, bias_size, 1, 1};
        int stride_bias[] = {bias_size, 1, 1, 1};
        if(cudnn_data_type_ == HIPDNN_DATA_FLOAT) {
            cudnn_check(hipdnnSetTensorNdDescriptor(bias_desc_, HIPDNN_DATA_FLOAT, 4, dim_bias, stride_bias));
        }
        else if(cudnn_data_type_ == HIPDNN_DATA_HALF) {
            cudnn_check(hipdnnSetTensorNdDescriptor(bias_desc_, HIPDNN_DATA_HALF, 4, dim_bias, stride_bias));
        } else {
            MNN_PRINT("only supports fp32/fp16 data type!!!\n");
        }
        use_bias_ = true;
    }
}

DeconvSingleInputExecution::~DeconvSingleInputExecution() {
    cudnn_check(hipdnnDestroyConvolutionDescriptor(conv_desc_));
    cudnn_check(hipdnnDestroyFilterDescriptor(filter_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(padded_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(output_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(input_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(bias_desc_));
    cudnn_check(hipdnnDestroyActivationDescriptor(act_desc_));

    if (nullptr != weightTensor) {
        backend()->onReleaseBuffer(weightTensor.get(), Backend::STATIC);
    }
    if(use_bias_ && nullptr != biasTensor) {
        backend()->onReleaseBuffer(biasTensor.get(), Backend::STATIC);
    }
    if(workspace_size_!=0 && nullptr != workspaceTensor) {
        backend()->onReleaseBuffer(workspaceTensor.get(), Backend::DYNAMIC_SEPERATE);
    }
}

ErrorCode DeconvSingleInputExecution::onResize(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    // prepare
    //MNN_PRINT("cuda DeconvSingleInput onResize in, pad:%d\n", mKernelInfo.padX);
    auto input = inputs[0], output = outputs[0];

    mIOInfo.iw = input->width();
    mIOInfo.ih = input->height();
    mIOInfo.ic = input->channel();
    mIOInfo.ib = input->batch();
    
    mIOInfo.ow = output->width();
    mIOInfo.oh = output->height();
    mIOInfo.oc = output->channel();
    mIOInfo.ob = output->batch();

    mKernelInfo.kernelN = output->channel();
    mKernelInfo.kernelC = input->channel() / mKernelInfo.groups;

    std::vector<int> in_shape = {mIOInfo.ib, mIOInfo.ic, mIOInfo.ih, mIOInfo.iw};
    std::vector<int> output_shape = {mIOInfo.ob, mIOInfo.oc, mIOInfo.oh, mIOInfo.ow};
    std::vector<int> filter_shape = {mKernelInfo.kernelC, mKernelInfo.kernelN, mKernelInfo.kernelY, mKernelInfo.kernelX};//deconv (ic oc kh kw)
    
    // printf("filter:%d %d %d %d\n", filter_shape[0], filter_shape[1], filter_shape[2], filter_shape[3]);
    // printf("input:%d %d %d %d\n", in_shape[0], in_shape[1], in_shape[2], in_shape[3]);
    // printf("output:%d %d %d %d\n", output_shape[0], output_shape[1], output_shape[2], output_shape[3]);
    cudnn_check(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, in_shape[0],
                                in_shape[1], in_shape[2], in_shape[3]));
 
    cudnn_check(hipdnnSetFilter4dDescriptor(filter_desc_, cudnn_data_type_, HIPDNN_TENSOR_NCHW, filter_shape[0],
                                filter_shape[1], filter_shape[2], filter_shape[3]));
    cudnn_check(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, output_shape[0],
                                output_shape[1], output_shape[2], output_shape[3]));

    

    hipdnnTensorDescriptor_t input_descriptor_real = nullptr;

    if (mKernelInfo.padMode == PadMode_SAME) {
        int kernelWidthSize = (mKernelInfo.kernelX - 1) * mKernelInfo.dilateX + 1;
        int kernelHeightSize = (mKernelInfo.kernelY - 1) * mKernelInfo.dilateY + 1;
        int pw = (mIOInfo.iw - 1) * mKernelInfo.strideX + kernelWidthSize - mIOInfo.ow;
        int ph = (mIOInfo.ih - 1) * mKernelInfo.strideY + kernelHeightSize - mIOInfo.oh;
        pad_left_  = pw/2;
        pad_right_ = pw - pad_left_;
        pad_top_ = ph/2;
        pad_bottom_ = ph - pad_top_;
    }

    use_pad_ = (pad_left_!=0 || pad_right_!=0 || pad_top_!=0 || pad_bottom_!=0 ) ? true : false;

    if(use_pad_) {
        int totalSize = output_shape[0]*output_shape[1]*(output_shape[2]+pad_top_+pad_bottom_)*(output_shape[3]+pad_left_+pad_right_);
        padTensor.reset(Tensor::createDevice<float>({totalSize}));
        backend()->onAcquireBuffer(padTensor.get(), Backend::DYNAMIC);
        mPadPtr = (void *)padTensor.get()->buffer().device;

        //dynamic memory release
        backend()->onReleaseBuffer(padTensor.get(), Backend::DYNAMIC);

        cudnn_check(hipdnnSetTensor4dDescriptor(padded_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, output_shape[0], output_shape[1],
            output_shape[2] + +pad_top_+pad_bottom_, output_shape[3] + pad_left_+pad_right_));
    }
    input_descriptor_real = use_pad_ ? padded_desc_ : input_desc_;

    cudnn_check(hipdnnSetConvolution2dDescriptor(conv_desc_, 0, 0, mKernelInfo.strideY, mKernelInfo.strideX, 
                                mKernelInfo.dilateY, mKernelInfo.dilateX, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    if (cudnn_data_type_ == HIPDNN_DATA_HALF) {
        cudnn_check(hipdnnSetConvolutionMathType(conv_desc_, HIPDNN_TENSOR_OP_MATH));
    }
    //set group num
    cudnn_check(hipdnnSetConvolutionGroupCount(conv_desc_, mKernelInfo.groups));
    
    // algorithm
    constexpr int requested_algo_count = 1;
    int returned_algo_count;
    hipdnnConvolutionBwdDataAlgoPerf_t perf_results;
    cudnn_check(cudnnGetConvolutionBackwardDataAlgorithm_v7(cudnn_handle_, filter_desc_, input_descriptor_real, conv_desc_,
        output_desc_,  requested_algo_count, &returned_algo_count, &perf_results));
    conv_bwd_algo_ = perf_results.algo;

    // workspace
    cudnn_check(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn_handle_, filter_desc_, input_descriptor_real, conv_desc_, output_desc_,
        conv_bwd_algo_, &workspace_size_));

    if (workspace_size_ != 0) {
        int workspaceSize = workspace_size_;
        workspaceTensor.reset(Tensor::createDevice<float>({workspaceSize}));
        //cudnn not support workspace memory reuse
        backend()->onAcquireBuffer(workspaceTensor.get(), Backend::DYNAMIC_SEPERATE);
        mWorkSpace = (void *)workspaceTensor.get()->buffer().device;
    }

    if(use_relu_) {
        cudnn_check(hipdnnSetActivationDescriptor(act_desc_, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
    } else if(use_relu6_) {
        cudnn_check(hipdnnSetActivationDescriptor(act_desc_, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, 6.0));
    } else {
        //do nothing
    }
    //MNN_PRINT("cuda DeconvSingleInput onResize out\n");
    return NO_ERROR;
}

ErrorCode DeconvSingleInputExecution::onExecute(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    //MNN_PRINT("cuda DeconvSingleInput onExecute in, inputsize:%d %d\n", (int)inputs.size(), workspace_size_);

    MNN_ASSERT(inputs.size() == 1);
    MNN_ASSERT(outputs.size() == 1);

    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    const void *input_addr = (const void*)inputs[0]->deviceId();
    const void *filter_addr = mFilter;
    const void *bias_addr = mBias;

    void *output_addr = (void*)outputs[0]->deviceId();
    void *workspace_addr = nullptr;
    if (workspace_size_ != 0) {
        workspace_addr = mWorkSpace;
    }

    const float alpha = 1;
    const float beta = 0;


    if(use_pad_) {
        cudnn_check(hipdnnConvolutionBackwardData(cudnn_handle_, &alpha, filter_desc_, filter_addr, input_desc_, input_addr, conv_desc_,
            conv_bwd_algo_, workspace_addr, workspace_size_, &beta, padded_desc_, mPadPtr));

        std::vector<int> out_shape = {mIOInfo.ob, mIOInfo.oc, mIOInfo.oh, mIOInfo.ow};

        int size = out_shape[0] * out_shape[1] * out_shape[2] * out_shape[3];
        int block_num = runtime->blocks_num(size);
        int threads_num = runtime->threads_num();

        cutPad<<<block_num, threads_num>>>(size, (float*)mPadPtr, out_shape[2]+pad_top_+pad_bottom_, out_shape[3]+pad_left_+pad_right_,
            out_shape[2], out_shape[3], pad_top_, pad_left_, (float*)output_addr);
    }
    else {
        cudnn_check(hipdnnConvolutionBackwardData(cudnn_handle_, &alpha, filter_desc_, filter_addr, input_desc_, input_addr, conv_desc_,
            conv_bwd_algo_, workspace_addr, workspace_size_, &beta, output_desc_, output_addr));
    }

    if(use_bias_) {
        cudnn_check(hipdnnAddTensor(cudnn_handle_, &alpha, bias_desc_, bias_addr, &alpha, output_desc_, output_addr));
    }
    if(use_relu_ || use_relu6_) {
        cudnn_check(hipdnnActivationForward(cudnn_handle_, act_desc_, &alpha, output_desc_, output_addr, &beta, output_desc_, output_addr));
    }
    return NO_ERROR;
}

class CUDADeconvolutionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs, 
            const MNN::Op* op, Backend* backend) const override {
        if (nullptr != op->main_as_Convolution2D()->quanParameter()) {
            auto quan = op->main_as_Convolution2D()->quanParameter();
            if (1 == quan->type() || 2 == quan->type()) {
                MNN_PRINT("cuda Deconv quant type 1 or 2 not support\n");
                return nullptr;
            }
        }

        if(inputs.size() == 3) {
            MNN_PRINT("Deconv inputs size:3 not support\n");
            return nullptr;
        } else if(inputs.size() == 1) {
            return new DeconvSingleInputExecution(backend, op);
        } else {
            MNN_PRINT("Deconv inputs size:%d not support", (int)inputs.size());
            return nullptr;
        }
    }
};

CUDACreatorRegister<CUDADeconvolutionCreator> __DeConvExecution(OpType_Deconvolution);

}// namespace CUDA
}// namespace MNN