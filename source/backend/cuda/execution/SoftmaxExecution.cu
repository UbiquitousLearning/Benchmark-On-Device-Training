#include "SoftmaxExecution.hpp"

namespace MNN {
namespace CUDA {

SoftmaxExecution::SoftmaxExecution(int axis, Backend *backend) : Execution(backend) {
    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    cudnn_handle_ = runtime->cudnn_handle();

    cudnn_check(hipdnnCreateTensorDescriptor(&input_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&output_desc_));

    cudnn_data_type_ = HIPDNN_DATA_FLOAT;
    mAxis = axis;
}

SoftmaxExecution::~SoftmaxExecution() {
    hipdnnDestroyTensorDescriptor(input_desc_);
    hipdnnDestroyTensorDescriptor(output_desc_);
}

ErrorCode SoftmaxExecution::onResize(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    inside = 1;
    outside = 1;
    if(mAxis < 0) {
        mAxis += inputs[0]->dimensions();
    }
    axis = inputs[0]->length(mAxis);
    for (int i=0; i<mAxis; ++i) {
        outside *= inputs[0]->length(i);
    }
    for (int i=mAxis+1; i<inputs[0]->dimensions(); ++i) {
        inside *= inputs[0]->length(i);
    }

    std::vector<int> tensor_shape = {outside, axis, inside, 1};
    cudnn_check(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, tensor_shape[0],
                                tensor_shape[1], tensor_shape[2], tensor_shape[3]));

    cudnn_check(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, tensor_shape[0],
                                tensor_shape[1], tensor_shape[2], tensor_shape[3]));

    return NO_ERROR;
}

ErrorCode SoftmaxExecution::onExecute(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto input = (void*)inputs[0]->deviceId();
    auto output = (void*)outputs[0]->deviceId();

    const float alpha = 1;
    const float beta = 0;
    cudnn_check(hipdnnSoftmaxForward(cudnn_handle_, HIPDNN_SOFTMAX_ACCURATE,
                HIPDNN_SOFTMAX_MODE_CHANNEL,
                &alpha,
                input_desc_, input,
                &beta,
                output_desc_, output));

    return NO_ERROR;
}

class SoftmaxCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                                const MNN::Op* op, Backend* backend) const override {
        auto type = inputs[0]->getType();
        if (type.code != halide_type_float) {
            MNN_PRINT("softmax data type:%s not support", type.code);
            return nullptr;
        }
        auto axis = op->main_as_Axis()->axis();
        return new SoftmaxExecution(axis, backend);
    }
};

static CUDACreatorRegister<SoftmaxCreator> __init(OpType_Softmax);
}
}