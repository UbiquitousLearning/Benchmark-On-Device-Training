#include "hip/hip_runtime.h"
#include "ConvDepthWiseExecution.hpp"
#include "core/ConvolutionCommon.hpp"
namespace MNN {
namespace CUDA {
struct constBuffer {
    int pad[2];
    int kernelSize[2];
    int stride[2];
    int dilate[2];
    int inputSize[2];
    int outputSize[2];
    int channel;
    int subChannel;
    int total;
    int activationType;
} uConstant;

ConvDepthWiseExecution::ConvDepthWiseExecution(const Op* op, Backend* bn) : Execution(bn) {
    mOp = op;
    auto pool = static_cast<CUDABackend*>(bn)->getStaticBufferPool();
    mConstBuffer = pool->alloc(sizeof(constBuffer));

    auto conv = mOp->main_as_Convolution2D();
    //weight host->device
    int weightSize = conv->weight()->size();
    weightTensor.reset(Tensor::createDevice<float>({weightSize}));
    backend()->onAcquireBuffer(weightTensor.get(), Backend::STATIC);
    mFilter = (void *)weightTensor.get()->buffer().device;
    cuda_check(hipMemcpy(mFilter, conv->weight()->data(), conv->weight()->size()*sizeof(float), hipMemcpyHostToDevice));

    mBias = nullptr;
    if(conv->bias()->size() != 0) {
        int biasSize = conv->bias()->size();
        biasTensor.reset(Tensor::createDevice<float>({biasSize}));
        backend()->onAcquireBuffer(biasTensor.get(), Backend::STATIC);
        mBias = (void *)biasTensor.get()->buffer().device;
        cuda_check(hipMemcpy(mBias, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));
        use_bias_ = true;
    }
}
ConvDepthWiseExecution::~ ConvDepthWiseExecution() {
    auto pool = static_cast<CUDABackend*>(backend())->getStaticBufferPool();
    pool->free(mConstBuffer);
    if (nullptr != weightTensor) {
        backend()->onReleaseBuffer(weightTensor.get(), Backend::STATIC);
    }
    if(use_bias_ && nullptr != biasTensor) {
        backend()->onReleaseBuffer(biasTensor.get(), Backend::STATIC);
    }
}

ErrorCode ConvDepthWiseExecution::onResize(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto pad = ConvolutionCommon::convolutionPad(inputs[0], outputs[0], mOp->main_as_Convolution2D()->common());
    auto conv = mOp->main_as_Convolution2D();
    auto convCommon = mOp->main_as_Convolution2D()->common();
    constBuffer parameters;
    parameters.pad[0] = pad.first;
    parameters.pad[1] = pad.second;
    parameters.kernelSize[0] = convCommon->kernelX();
    parameters.kernelSize[1] = convCommon->kernelY();
    parameters.stride[0] = convCommon->strideX();
    parameters.stride[1] = convCommon->strideY();
    parameters.dilate[0] = convCommon->dilateX();
    parameters.dilate[1] = convCommon->dilateY();
    parameters.inputSize[0] = inputs[0]->width();
    parameters.inputSize[1] = inputs[0]->height();
    parameters.channel = inputs[0]->batch() * inputs[0]->channel();
    parameters.outputSize[0] = outputs[0]->width();
    parameters.outputSize[1] = outputs[0]->height();
    parameters.total = parameters.channel * parameters.outputSize[1] * parameters.outputSize[0];
    parameters.subChannel = inputs[0]->channel();
    parameters.activationType = convCommon->relu() ? 1 : (convCommon->relu6() ? 2 : 0);

    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    runtime->memcpy((uint8_t*)mConstBuffer.first + mConstBuffer.second, &parameters, sizeof(constBuffer), MNNMemcpyHostToDevice);
    mTotalCount = parameters.total;

    return NO_ERROR;
}

__global__ void CONV_DW(const float* input, const float* kernel, const float* bias, float *output, const constBuffer* uConstant) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < uConstant->total; i += blockDim.x * gridDim.x) {
        {
            int iw = uConstant->inputSize[0];
            int ih = uConstant->inputSize[1];
            int c = uConstant->channel;
            int ow = uConstant->outputSize[0];
            int oh = uConstant->outputSize[1];
            int kw = uConstant->kernelSize[0];
            int kh = uConstant->kernelSize[1];
            int dw = uConstant->dilate[0];
            int dh = uConstant->dilate[1];
            int sw = uConstant->stride[0];
            int sh = uConstant->stride[1];
            int pw = uConstant->pad[0];
            int ph = uConstant->pad[1];
            int acttype = uConstant->activationType;

            int oz = i / (ow * oh);
            int tmp = i % (ow * oh);
            int oy = tmp / ow;
            int ox = tmp % ow;
            int kz = oz % uConstant->subChannel;
            
            int ix = ox * sw - pw;
            int iy = oy * sh - ph;
            float color = 0.0;
            if (bias != nullptr) {
                color = bias[kz];
            }

            int fx, fy, fz;
            for (fy=0; fy<kh; ++fy) {
                int sy = fy*dh + iy;
                if (sy >= ih || sy < 0) {
                    continue;
                }
                for (fx=0; fx<kw; ++fx) {
                    int sx = fx*dw + ix;
                    if (sx >= iw || sx < 0) {
                        continue;
                    }
                    float inputValue = input[0
                        + sx
                        + sy * iw
                        + oz * iw * ih
                    ];
                    float k = kernel[0
                        + fx
                        + fy * kw
                        + kz * kw * kh
                    ];
                    color  += k*inputValue;
                }
            }
            color = (acttype==1) ? max(0.0, color) : (acttype==2 ? (min(max(0.0, color), 6.0)) : color);
            output[0
                + ox
                + oy * ow
                + oz * ow * oh
            ] = color;
        }
    }
    return;
}


ErrorCode ConvDepthWiseExecution::onExecute(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    int block_num = runtime->blocks_num(mTotalCount);
    int threads_num = runtime->threads_num();
    auto constPtr = (uint8_t*)mConstBuffer.first + mConstBuffer.second;
    if (inputs.size() == 1) {
        CONV_DW<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const float*)mFilter,
             (const float*)mBias, (float*)outputs[0]->deviceId(), (const constBuffer*)(constPtr));
    } else if (inputs.size() == 3) {
        CONV_DW<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const float*)inputs[1]->deviceId(),
             (const float*)inputs[2]->deviceId(), (float*)outputs[0]->deviceId(), (const constBuffer*)constPtr);
    } else {
        MNN_ASSERT(inputs.size() == 2);
        CONV_DW<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const float*)inputs[1]->deviceId(),
             nullptr, (float*)outputs[0]->deviceId(), (const constBuffer*)constPtr);
    }
    return NO_ERROR;
}



__global__ void DECONV_DW(const float* input, const float* kernel, const float* bias, float *output, const constBuffer* uConstant) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < uConstant->total; i += blockDim.x * gridDim.x) {
        {
            int iw = uConstant->inputSize[0];
            int ih = uConstant->inputSize[1];
            int c = uConstant->channel;
            int ow = uConstant->outputSize[0];
            int oh = uConstant->outputSize[1];
            int kw = uConstant->kernelSize[0];
            int kh = uConstant->kernelSize[1];
            int dw = uConstant->dilate[0];
            int dh = uConstant->dilate[1];
            int sw = uConstant->stride[0];
            int sh = uConstant->stride[1];
            int pw = uConstant->pad[0];
            int ph = uConstant->pad[1];

            int oz = i / (ow * oh);
            int tmp = i % (ow * oh);
            int oy = tmp / ow;
            int ox = tmp % ow;
            int kz = oz % uConstant->subChannel;
            
            int ix = ox + pw;
            int iy = oy + ph;
            float color = 0.0;
            if (bias != nullptr) {
                color = bias[kz];
            }

            int fx, fy, fz;
            for (fy=0; fy<kh; ++fy) {
                int sy = iy - fy*dh;
                int y = sy / sh;
                if (sy % sh == 0 && y >= 0 && y < ih) {
                    for (int fx=0; fx<kw; ++fx) {
                        int sx = ix - fx*dw;
                        int x = sx / sw;
                        if (sx % sw == 0 && x >= 0 && x < iw) {
                            float inputValue = input[0
                                + x
                                + y * iw
                                + oz * iw * ih
                            ];
                            float k = kernel[0
                                + fx
                                + fy * kw
                                + kz * kw * kh
                            ];
                            color  += k*inputValue;                            
                        }
                    }
                }
            }
            output[0
                + ox
                + oy * ow
                + oz * ow * oh
            ] = color;
        }
    }
    return;
}


ErrorCode DeconvDepthWiseExecution::onResize(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto convCommon = mOp->main_as_Convolution2D()->common();
    auto pad = ConvolutionCommon::convolutionTransposePad(inputs[0], outputs[0], convCommon);
    constBuffer parameters;
    parameters.pad[0] = pad.first;
    parameters.pad[1] = pad.second;
    parameters.kernelSize[0] = convCommon->kernelX();
    parameters.kernelSize[1] = convCommon->kernelY();
    parameters.stride[0] = convCommon->strideX();
    parameters.stride[1] = convCommon->strideY();
    parameters.dilate[0] = convCommon->dilateX();
    parameters.dilate[1] = convCommon->dilateY();
    parameters.inputSize[0] = inputs[0]->width();
    parameters.inputSize[1] = inputs[0]->height();
    parameters.channel = inputs[0]->batch() * inputs[0]->channel();
    parameters.outputSize[0] = outputs[0]->width();
    parameters.outputSize[1] = outputs[0]->height();
    parameters.total = parameters.channel * parameters.outputSize[1] * parameters.outputSize[0];
    parameters.subChannel = inputs[0]->channel();
    auto constPtr = (uint8_t*)mConstBuffer.first + mConstBuffer.second;

    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    runtime->memcpy(constPtr, &parameters, sizeof(constBuffer), MNNMemcpyHostToDevice);
    mTotalCount = parameters.total;
    return NO_ERROR;
}

ErrorCode DeconvDepthWiseExecution::onExecute(const std::vector<Tensor *> &inputs, const std::vector<Tensor *> &outputs) {
    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    int block_num = runtime->blocks_num(mTotalCount);
    int threads_num = runtime->threads_num();
    auto constPtr = (uint8_t*)mConstBuffer.first + mConstBuffer.second;
    if (inputs.size() > 2) {
        DECONV_DW<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const float*)inputs[1]->deviceId(),
             (const float*)inputs[2]->deviceId(), (float*)outputs[0]->deviceId(), (const constBuffer*)constPtr);
    } else {
        DECONV_DW<<<block_num, threads_num>>>((const float*)inputs[0]->deviceId(), (const float*)inputs[1]->deviceId(),
             nullptr, (float*)outputs[0]->deviceId(), (const constBuffer*)constPtr);
    }
    return NO_ERROR;
}


class ConvDepthWiseExecutionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs,
                                const MNN::Op* op, Backend* backend) const override {
        if (OpType_ConvolutionDepthwise == op->type()) {
            return new ConvDepthWiseExecution(op, backend);
        }
        if (inputs.size() == 1) {
            MNN_PRINT("deconv depthwise not support 1 input yet\n");
            return nullptr;
        }
        return new DeconvDepthWiseExecution(op, backend);
    }
};

static CUDACreatorRegister<ConvDepthWiseExecutionCreator> __init(OpType_ConvolutionDepthwise);
static CUDACreatorRegister<ConvDepthWiseExecutionCreator> __init2(OpType_DeconvolutionDepthwise);
}
}